#include "hip/hip_runtime.h"
# include "graph.h"
# include <opencv2/opencv.hpp>

__device__ float intersect(const Object& obj, const vec3& origin, const vec3& dir){
    switch(obj.type){
        case SPHERE:

            const vec3 OC = obj.position - origin;

            if (glm::length(OC) < obj.radius || glm::dot(OC, dir) < 0) return infinity;

            float l        = glm::length(glm::dot(OC, dir));
            float m_square = glm::length(OC) * glm::length(OC) - l * l;
            float q_square = obj.radius * obj.radius - m_square;

            return (q_square >= 0) ? (l - sqrt(q_square)) : infinity;

        case PLANE:
            float dn = glm::dot(dir, obj.normal);
    
            if (abs(dn) < 1e-6) { return infinity; }
            
            float d = glm::dot(obj.position - origin, obj.normal) / dn;
            
            return d > 0 ? d : infinity;
    }
}

__device__ vec3 get_normal(const Object& obj, const vec3& point){
    switch (obj.type) {
        case SPHERE:
            return glm::normalize(point - obj.position);
        case PLANE:
            return obj.normal;
    }
}

__device__ vec3 intersect_color(
    const vec3 origin, const vec3 dir, 
    const float intensity, 
    const Object* dev_scene, const int numObjects
){

    float min_distance = infinity;

    size_t obj_index=-1;
    for (size_t i = 0; i < numObjects; ++i) {
        float current_distance = intersect(dev_scene[obj_index], origin, dir);
        if (current_distance < min_distance) {
            min_distance = current_distance;
            obj_index = i;
        }
    }

    if (min_distance == infinity || intensity < 0.01) return vec3(0., 0., 0.);
    
    Object* obj   = dev_scene[obj_index];
    const vec3 P  = origin + dir * min_distance;
    const vec3 PL = glm::normalize(light_point - P);
    const vec3 PO = glm::normalize(origin - P);
    const vec3 N  = get_normal(obj, P);

    vec3 c = ambient * obj.color;

    /*shadow test*/
    float l[maxObjects];
    int lSize = 0;
    for (size_t i = 0; i < numObjects; ++i) {
        if (i != obj_index){
            float intersection = intersect(dev_scene[i], P + N * .0001f, PL);
            if (intersection < glm::length(PL)) {
                l[lSize++] = intersection;
            }
        }
    }

    bool isShadowed = false;
    for (size_t i = 0; i < lSize; ++i) {
        if (l[i] < glm::length(PL)) {
            isShadowed = true;
            break;
        }
    }

    if (!isShadowed) {
        c += obj.diffuse * std::max(glm::dot(N, PL), 0.f) * obj.color * light_color;
        c += obj.specular_coef * powf(std::max(glm::dot(N, glm::normalize(PL + PO)), 0.f), obj.specular_k) * light_color;
    }

    vec3 reflect_ray = dir - 2 * glm::dot(dir, N) * N;
    c += obj.reflection * intersect_color(P + N * .0001f, reflect_ray, obj.reflection * intensity, dev_scene, numObjects);
    return glm::clamp(c, 0.f, 1.f);
}

__global__ void rendering_kerenl(
    const float lowerX, const float lowerY,
    const float upperX, const float upperY,
    const float stepX, const float stepY,
    const int w, const int h, 
    const Object* dev_scene, vec3* gpu_output,
    const vec3 camera_dir, const vec3 camera_right, const vec3 camera_up
){

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX >= w || thisY >= h) return;

    float u = upperX - thisX * stepX;
    float v = upperY - thisY * stepY;

    vec3 direction = glm::normalize(camera_dir + u * camera_right + v * camera_up);

    int index = thisY * w + thisX;
    gpu_output[index] = intersect_color(camera_pos, direction, 1, dev_scene);
}

void rendering(
    const int w, const int h,
    const Object* dev_scene, const int numObjects,
    const std::string filename
){
    
    const float r     = float(w) / h;                                    // aspect ratio
    const glm::vec4 S = glm::vec4(-1., -1. / r + .25, 1., 1. / r + .25); // view frustum

    const vec3 camera_dir   = glm::normalize(camera_target - camera_pos);
    const vec3 camera_right = glm::normalize(glm::cross(camera_dir, vec3(0, 1, 0)));
    const vec3 camera_up    = glm::normalize(glm::cross(camera_right, camera_dir)); 

    const float stepX = (S.z - S.x) / (w - 1);
    const float stepY = (S.w - S.y) / (h - 1);

    size_t outputSize = w * h * sizeof(vec3);
    vec3 *gpu_output; // dev output
    hipMalloc(&gpu_output, outputSize);

    dim3 blockSize(16, 16);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);

    rendering_kerenl<<<gridSize, blockSize>>>(
        S.x, S.y, 
        S.z, S.w, 
        stepX, stepY, 
        w, h, 
        dev_scene, gpu_output,
        const vec3 camera_dir, const vec3 camera_right, const vec3 camera_up
    );

    cv::Mat img(h, w, CV_32FC3);
    hipMemcpy(img.data, gpu_output, outputSize, hipMemcpyDeviceToHost);

    img *= 255;
    img.convertTo(img, CV_8UC3);
    cv::imwrite(filename, img);

    hipFree(gpu_output);
}

// /* Sphere */
// __device__ float intersect_sphere(const Object& obj, const vec3& origin, const vec3& dir){}
// __device__ vec3 get_normal_sphere(const Object& obj, const vec3& point){}
// /* Plane */
// __device__ float intersect_plane(const Object& obj, const vec3& origin, const vec3& dir){}
// __device__ vec3 get_normal_plane(const Object& obj, const vec3& point){}
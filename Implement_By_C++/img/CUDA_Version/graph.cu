#include "hip/hip_runtime.h"
# include "graph.h"

# include <iostream>
# include <chrono> 

class CudaTimer {
public:
    CudaTimer() {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_);
    }
    ~CudaTimer() {
        hipEventRecord(stop_);
        hipEventSynchronize(stop_);
        float ms = 0;
        hipEventElapsedTime(&ms, start_, stop_);
        std::cout << "[Kernel] " << " took " << ms << " ms" << std::endl;
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }
private:
    hipEvent_t start_, stop_;
};

__device__ int rendering_balanced_dev_cnt=0;

__device__ float intersect(const Object& obj, const vec3& origin, const vec3& dir){
    switch(obj.type){
        case SPHERE:{
            const vec3 OC = obj.position - origin;

            if (glm::length(OC) < obj.radius || glm::dot(OC, dir) < 0) return infinity;

            float l        = glm::length(glm::dot(OC, dir));
            float m_square = glm::length(OC) * glm::length(OC) - l * l;
            float q_square = obj.radius * obj.radius - m_square;

            return (q_square >= 0) ? (l - sqrtf(q_square)) : infinity;
        }
        case PLANE: {
            float dn = glm::dot(dir, obj.normal);
    
            if (abs(dn) < 1e-6) { return infinity; }
            
            float d = glm::dot(obj.position - origin, obj.normal) / dn;
            
            return d > 0 ? d : infinity;
        }
    }
}

__device__ vec3 get_normal(const Object& obj, const vec3& point){
    switch (obj.type) {
        case SPHERE:
            return glm::normalize(point - obj.position);
        case PLANE:
            return obj.normal;
    }
}

__device__ vec3 intersect_color(
    vec3 origin, vec3 dir, 
    const float initial_intensity, 
    const Object* dev_scene
){
    vec3 final_color = vec3(0., 0., 0.);
    float intensity = initial_intensity;

    for (int depth = 0; depth < MAX_DEPTH; ++depth) {
        if (intensity < 0.01) break;

        float min_distance = infinity;
        size_t obj_index = invalid_idx;
        for (size_t i = 0; i < numObjects; ++i) {
            float current_distance = intersect(dev_scene[i], origin, dir);
            if (current_distance < min_distance) {
                min_distance = current_distance;
                obj_index = i;
            }
        }

        if (min_distance == infinity) break;
        
        const Object& obj = dev_scene[obj_index];
        vec3 c            = ambient * obj.color;
        const vec3 P      = origin + dir * min_distance;
        const vec3 PL     = glm::normalize(light_point - P);
        const vec3 PO     = glm::normalize(origin - P);
        const vec3 N      = get_normal(obj, P);

        /*shadow test*/
        bool in_shadow = false;
        for (size_t i = 0; i < numObjects; ++i) {
            if (i != obj_index){
                float intersection = intersect(dev_scene[i], P + N * .0001f, PL);
                if (intersection < glm::length(PL)){
                    in_shadow = true;
                    break;
                }
            }
        }

        if (!in_shadow) {
            c += obj.diffuse * fmaxf(glm::dot(N, PL), 0.f) * obj.color * light_color;
            c += obj.specular_coef * powf(fmaxf(glm::dot(N, glm::normalize(PL + PO)), 0.f), obj.specular_k) * light_color;
        }

        final_color += intensity * c;
        if (obj.reflection <= 0) break;

        dir = dir - 2 * glm::dot(dir, N) * N;
        origin = P + N * .0001f;
        intensity *= obj.reflection;
    }

    return glm::clamp(final_color, 0.f, 1.f);
}

__global__ void rendering_kernel(
    const float lowerX, const float lowerY,
    const float upperX, const float upperY,
    const float stepX, const float stepY,
    const int w, const int h, 
    const Object* dev_scene, vec3* gpu_output,
    const vec3 camera_dir, const vec3 camera_right, const vec3 camera_up
){

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX >= w || thisY >= h) return;

    float u = upperX - thisX * stepX;
    float v = upperY - thisY * stepY;

    vec3 direction = glm::normalize(camera_dir + u * camera_right + v * camera_up);

    int index = thisY * w + thisX;
    gpu_output[index] = intersect_color(camera_pos, direction, 1, dev_scene);
}

void rendering(
    const int w, const int h,
    const std::string filename
){
   
    const float r     = float(w) / h;                                    // aspect ratio
    const glm::vec4 S = glm::vec4(-1., -1. / r + .25, 1., 1. / r + .25); // view frustum

    const vec3 camera_dir   = glm::normalize(camera_target - camera_pos);
    const vec3 camera_right = glm::normalize(glm::cross(camera_dir, vec3(0, 1, 0)));
    const vec3 camera_up    = glm::normalize(glm::cross(camera_right, camera_dir)); 

    const float stepX = (S.z - S.x) / (w - 1);
    const float stepY = (S.w - S.y) / (h - 1);

    /* setup dev_scene */
    Object *dev_scene;
    hipMalloc(&dev_scene, sizeof(host_scene));
    hipMemcpy(dev_scene, host_scene, sizeof(host_scene), hipMemcpyHostToDevice);

    /* setup dev_output */
    size_t outputSize = w * h * sizeof(vec3);
    vec3 *gpu_output; // dev output
    hipMalloc(&gpu_output, outputSize);

    dim3 blockSize(16, 16);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);

    {
        CudaTimer timer;
        rendering_kernel<<<gridSize, blockSize>>>(
            S.x, S.y, 
            S.z, S.w, 
            stepX, stepY, 
            w, h, 
            dev_scene, gpu_output,
            camera_dir, camera_right, camera_up
        );
    }

    cv::Mat img(h, w, CV_32FC3);
    hipMemcpy(img.data, gpu_output, outputSize, hipMemcpyDeviceToHost);

    img *= 255;
    img.convertTo(img, CV_8UC3);
    cv::imwrite(filename, img);

    hipFree(dev_scene);
    hipFree(gpu_output);
}

__global__ void rendering_kernel_balanced(
    const float lowerX, const float lowerY,
    const float upperX, const float upperY,
    const float stepX, const float stepY,
    const int w, const int h, 
    const Object* dev_scene, vec3* gpu_output,
    const vec3 camera_dir, const vec3 camera_right, const vec3 camera_up
){
    // <<<1d grid, 1d block>>>
    while (true) {
        int index = atomicAdd(&rendering_balanced_dev_cnt, 1);
        if (index>w*h) break;

        int i=index/w, j=index%w;
        float u = upperX - j * stepX;
        float v = upperY - i * stepY;
        vec3 direction = glm::normalize(camera_dir + u * camera_right + v * camera_up);
        gpu_output[index] = intersect_color(camera_pos, direction, 1, dev_scene);
    }
}

void rendering_balanced (
    const int w, const int h,
    const std::string filename
) {
    const float r     = float(w) / h;                                    // aspect ratio
    const glm::vec4 S = glm::vec4(-1., -1. / r + .25, 1., 1. / r + .25); // view frustum

    const vec3 camera_dir   = glm::normalize(camera_target - camera_pos);
    const vec3 camera_right = glm::normalize(glm::cross(camera_dir, vec3(0, 1, 0)));
    const vec3 camera_up    = glm::normalize(glm::cross(camera_right, camera_dir)); 

    const float stepX = (S.z - S.x) / (w - 1);
    const float stepY = (S.w - S.y) / (h - 1);

    /* setup dev_scene */
    Object *dev_scene;
    hipMalloc(&dev_scene, sizeof(host_scene));
    hipMemcpy(dev_scene, host_scene, sizeof(host_scene), hipMemcpyHostToDevice);

    /* setup dev_output */
    size_t outputSize = w * h * sizeof(vec3);
    vec3 *gpu_output; // dev output
    hipMalloc(&gpu_output, outputSize);

    dim3 blockSize(256);
    dim3 gridSize((w*h+blockSize.x-1)/blockSize.x);

    constexpr int _zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(rendering_balanced_dev_cnt), &_zero, sizeof(int));

    {
        CudaTimer timer;
        rendering_kernel_balanced<<<gridSize, blockSize>>>(
            S.x, S.y, 
            S.z, S.w, 
            stepX, stepY, 
            w, h, 
            dev_scene, gpu_output,
            camera_dir, camera_right, camera_up
        );
    }

    cv::Mat img(h, w, CV_32FC3);
    hipMemcpy(img.data, gpu_output, outputSize, hipMemcpyDeviceToHost);

    img *= 255;
    img.convertTo(img, CV_8UC3);
    cv::imwrite(filename, img);  

    hipFree(dev_scene);
    hipFree(gpu_output);
}